#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ float schlickFresnel(float cosTheta, float ior) {
    float r0 = (1.0f - ior) / (1.0f + ior);
    r0 = r0 * r0;
    return r0 + (1.0f - r0) * pow(1.0f - cosTheta, 5.0f);
}

__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    
    if (m.hasRefractive > 0.0f) {
        // Refractive material (glass)
        glm::vec3 incident = pathSegment.ray.direction;
        float cosTheta = glm::dot(-incident, normal);
        bool entering = cosTheta > 0;
        
        float eta = entering ? 1.0f / m.indexOfRefraction : m.indexOfRefraction;
        glm::vec3 n = entering ? normal : -normal;
        cosTheta = abs(cosTheta);
        
        // Fresnel reflection probability (use material IOR, not eta)
        float fresnel = schlickFresnel(cosTheta, m.indexOfRefraction);
        
        if (u01(rng) < fresnel) {
            // Reflection
            pathSegment.ray.direction = glm::reflect(incident, n);
            pathSegment.ray.origin = intersect + 0.001f * n;
        } else {
            // Refraction
            glm::vec3 refracted = glm::refract(incident, n, eta);
            if (glm::length(refracted) < 0.001f) {
                // Total internal reflection
                pathSegment.ray.direction = glm::reflect(incident, n);
                pathSegment.ray.origin = intersect + 0.001f * n;
            } else {
                pathSegment.ray.direction = refracted;
                pathSegment.ray.origin = intersect - 0.001f * n;
            }
        }
    } else if (m.hasReflective > 0.0f) {
        // Perfect specular reflection
        pathSegment.ray.direction = glm::reflect(pathSegment.ray.direction, normal);
        pathSegment.ray.origin = intersect + 0.001f * normal;
    } else {
        // Diffuse material
        pathSegment.ray.direction = calculateRandomDirectionInHemisphere(normal, rng);
        pathSegment.ray.origin = intersect + 0.001f * normal;
    }
    
    pathSegment.color *= m.color;
}